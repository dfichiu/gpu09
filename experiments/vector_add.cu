#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int n = 1 << 20; // Vector size: 2^20 = 1,048,576 elements
    size_t bytes = n * sizeof(float);

    // Allocate host memory
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);

    // Initialize input vectors
    for (int i = 0; i < n; i++) {
        h_A[i] = i * 0.5f;
        h_B[i] = i * 0.3f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, bytes);
    hipMalloc((void**)&d_B, bytes);
    hipMalloc((void**)&d_C, bytes);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Set up execution configuration
    int threads = 256;
    int blocks = (n + threads - 1) / threads;

    // Start GPU computation and time it
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    vectorAdd<<<blocks, threads>>>(d_A, d_B, d_C, n);
    hipEventRecord(stop);

    // Wait for GPU to finish and calculate elapsed time
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy result back to host
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    printf("Vector addition completed in %.3f ms\n", milliseconds);

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

