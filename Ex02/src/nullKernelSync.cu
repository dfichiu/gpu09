#include "hip/hip_runtime.h"
/*
 *
 * nullKernelAsync.cu
 *
 * Microbenchmark for throughput of asynchronous kernel launch.
 *
 * Build with: nvcc -I ../chLib <options> nullKernelAsync.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include "chTimer.h"

__global__
void
NullKernel()
{
}

int
main()
{
    const int cIterations = 1000000;

    int numBlocks = 16384;
    int threadsPerBlock = 1024;

    chTimerTimestamp start, stop;

    printf( "numBlocks,threadsPerBlock,usPerLaunch\n" ); fflush( stdout );

    while ( numBlocks > 0) {
        threadsPerBlock = 1024;
        while ( threadsPerBlock > 1 ) {
            chTimerGetTime( &start );
            for ( int i = 0; i < cIterations; i++ ) {
                NullKernel<<<numBlocks,threadsPerBlock>>>();
                hipDeviceSynchronize();
            }
            chTimerGetTime( &stop );

            {
                double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
                double usPerLaunch = microseconds / (float) cIterations;

                printf( "%d,%d,%.2f us\n", numBlocks, threadsPerBlock, usPerLaunch );
                fflush( stdout );
            }

            threadsPerBlock = threadsPerBlock / 2;
        }
        
        numBlocks = numBlocks - 1024;
    }
    
    return 0;
}
