#include "hip/hip_runtime.h"
/*
 *
 * nullKernelAsync.cu
 *
 * Microbenchmark for throughput of asynchronous kernel launch.
 *
 * Build with: nvcc -I ../chLib <options> nullKernelAsync.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>

#include "chTimer.h"

__device__ long long int clockDiff;

__global__
void
busyWaitKernel(long long int cyclesToBusyWait, bool writeToVariable)
{
    long long int start = clock64();

    while ( clock64() -  start < cyclesToBusyWait ) {
        // Busy-wait
    }

    long long int end = clock64();

    if ( writeToVariable ) {
        clockDiff = end - start;
    }
}

int
main()
{
    const int cIterations = 1000000;

    long long int cyclesToBusyWait = 0;
    bool writeToVariable = true;
    bool run = true;

    chTimerTimestamp start, stop;

    printf( "cyclesToBusyWait,usPerLaunch\n" ); fflush( stdout );

    while (run) {
        chTimerGetTime( &start );
        for ( int i = 0; i < cIterations; i++ ) {
            busyWaitKernel<<<1,1>>>(cyclesToBusyWait, writeToVariable);
        }
        hipDeviceSynchronize();
        chTimerGetTime( &stop );

        {
            double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
            double usPerLaunch = microseconds / (float) cIterations;

            printf( "%llu,%.2f us\n", cyclesToBusyWait, usPerLaunch );
            fflush( stdout );
        }
        cyclesToBusyWait = cyclesToBusyWait + 100;
    }   

return 0;
}
