#include "hip/hip_runtime.h"
/*************************************************************************************************
 *
 *        Computer Engineering Group, Heidelberg University - GPU Computing Exercise 03
 *
 *                           Group : TBD
 *
 *                            File : main.cu
 *
 *                         Purpose : Memory Operations Benchmark
 *
 *************************************************************************************************/

//
// Kernels
//

__global__ void 
globalMemCoalescedKernel( int d_memoryA[N] ,int d_memoryB[N], int memSize)
{
   int indx = blockIdx.x * blockDim.x + threadIdx.x;
   if (indx + memSize - 1 < N){
    for (int i=0; i<memSize; i++) {
        d_memoryB[indx+i] = d_memoryA[indx+i];
        }
   }
}

void 
globalMemCoalescedKernel_Wrapper(dim3 gridDim, dim3 blockDim, int* d_memoryA ,int* d_memoryB , int memSize ) {
	globalMemCoalescedKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>( d_memoryA ,d_memoryB,  memSize);
}

__global__ void 
globalMemStrideKernel(/*TODO Parameters*/)
{
    return  ;/*TODO Kernel Code*/
}

void 
globalMemStrideKernel_Wrapper(dim3 gridDim, dim3 blockDim /*TODO Parameters*/) {
	globalMemStrideKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>( /*TODO Parameters*/);
}

__global__ void 
globalMemOffsetKernel(/*TODO Parameters*/)
{
    return  ;/*TODO Kernel Code*/
}

void 
globalMemOffsetKernel_Wrapper(dim3 gridDim, dim3 blockDim /*TODO Parameters*/) {
	globalMemOffsetKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>( /*TODO Parameters*/);
}

